#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <hip/hip_fp16.h>
#define StartAxis(i,axis) int i = blockIdx.axis * blockDim.axis + threadIdx.axis;
#define GRID_LOOP_X(i, n)                                 \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
         i += blockDim.x * gridDim.x)

#define GRID_AXIS_LOOP(i, n, axis)                                 \
    for (int i = blockIdx.axis * blockDim.axis + threadIdx.axis; i < n; \
         i += blockDim.axis * gridDim.axis)





__device__ __half2  h2agtb(__half2 a, __half2 b, __half gtval, __half leval ){
    if (__hbgt2(a,b)){
        return __halves2half2(gtval,gtval);
    } 
        return __halves2half2(__hgt(__low2half(a),__low2half(b)) ? gtval : leval,
                              __hgt(__high2half(a),__high2half(b)) ? gtval : leval); 
    
 }
__device__ __half2  h2ageb(__half2 a, __half2 b, __half geval, __half ltval ){
  
    if (__hbge2(a,b)){
    return __halves2half2(geval,geval);
    }
    return __halves2half2(__hge(__low2half(a),__low2half(b)) ? geval : ltval,
                          __hge(__high2half(a),__high2half(b)) ? geval : ltval);
  
}
__device__ __half2  h2altb(__half2 a, __half2 b, __half geval, __half ltval ){
    if (__hblt2(a,b)){
    return __halves2half2(ltval,ltval);
    }
    return __halves2half2(__hlt(__low2half(a),__low2half(b)) ?ltval: geval,
                          __hlt(__high2half(a),__high2half(b)) ?ltval: geval);
  }
__device__ __half2  h2aleb(__half2 a, __half2 b, __half gtval, __half leval ){
    if (__hble2(a,b)){
    return __halves2half2(leval,leval);
    }
    return __halves2half2(__hle(__low2half(a),__low2half(b)) ?leval: gtval,
                          __hle(__high2half(a),__high2half(b)) ?leval: gtval);
}

extern "C" __global__ void Transpose(int numthreads,
               const float *src,
               const int *buf,
               const int ndims,
               float *dest)
{
    const int *src_strides = buf; 
    const int *dest_strides = &buf[ndims];
    const int *perm = &buf[ndims * 2];

    GRID_LOOP_X(destIdx, numthreads)
    {
        int srcIdx = 0;
        int t = destIdx;
        for (int i = 0; i < ndims; ++i)
        {
            const int ratio = t / dest_strides[i];
            t -= ratio * dest_strides[i];
            srcIdx += (ratio * src_strides[perm[i]]);
        }
        dest[destIdx] = src[srcIdx];
    }  
}



/*SwapEveryOther will swap the batches between 2 tensors. 
 It will be either the even or the odd.
   Both tensors have to be equal in size and dims.
   if even is >0 then it will do the even batches.
   Make sure labels are swapped on host end.
   */
extern "C" __global__ void SwapEveryOther(
    const int xThreads, //total batches
    const int totalbatches,
    float *t1,
    float *t2,
   const int start,
const int stride)
{
const int BVol = xThreads;

            for (int i =start;i<totalbatches;i+=stride)
        {   
                GRID_LOOP_X(xIdx, xThreads)
                { 
                    const float swapper =  t1[(i*BVol)+(xIdx)];
                    t1[(i*BVol) +xIdx]=t2[(i*BVol)+xIdx];
                    t2[(i*BVol)+xIdx]=swapper;
                }

            __syncthreads();
        }    
}



//SwapUpperLower will swap either the upper or lower batches
//Right Now inverse doesn't do anything
extern "C" __global__ void SwapUpperLower(
    const int xThreads, //batchsize
    const int yThreads, //batchvol
    float *t1,
    float *t2,
    const int t1upper,
    const int t2upper,
    const int inverse)
{
const int BVol = yThreads;
  
    if (t1upper>0)
    {
        GRID_AXIS_LOOP(xIdx, xThreads/2,x)
        { 
            int t2Idx;
            if (t2upper>0){
                t2Idx=xIdx;
            }else{
                t2Idx=xThreads/2 +xIdx;
            }
           
            if (xIdx < xThreads && t2Idx<xThreads)
            {
                GRID_AXIS_LOOP(yIdx, yThreads,y)
                {
                    
                    const float swapper =  t1[(xIdx*BVol)+(yIdx)];
                    t1[(xIdx*BVol) +yIdx]=t2[(t2Idx*BVol)+yIdx];
                    t2[(xIdx*BVol)+yIdx]=swapper;
                } 
            }
        }   
    }
    else  
    {
        GRID_AXIS_LOOP(xIdx, xThreads/2,x)
        {
            const int halfIdx=(xThreads/2)+xIdx;
            int t2Idx;
            if (t2upper>0){
                t2Idx=xIdx;
            }else{
                t2Idx=halfIdx;
            }
         
            if (halfIdx < xThreads)
            {
                GRID_AXIS_LOOP(yIdx, yThreads,y)
                {
                    const float swapper =  t1[(halfIdx*BVol)+(yIdx)];
                    t1[(halfIdx*BVol) +yIdx]=t2[(t2Idx*BVol)+yIdx];
                    t2[(halfIdx*BVol)+yIdx]=swapper;
                }
            }
        }   
    }
}

   
//ShapetoBatch4DNHWC Does a stride shape to batch. Make sure values on receiving end are set to zero when s2b is 0
extern "C" __global__ void ShapetoBatch4DNHWC(
    const int xThreads,
    const int yThreads,
    const int zThreads,
    const int hSize,
    const int wSize,
    const int num_original_batches,
    const int BatchVolume,
    const int OriginalVol,
    const int N1,
    const int N2,
    const int hstride,
    const int wstride,
    float *shape,
    float *batch,
    const int h_over_scan,
    const int w_over_scan,
    const bool S2B)
{
    int batch0 = N2 * xThreads * yThreads * zThreads;
    int batch1 = xThreads * yThreads * zThreads;
    int batch2 = yThreads * zThreads;
    int batch3 = zThreads;
    for (int b = 0;b<num_original_batches;b++)
    {
        const int ShapeOffset = OriginalVol*b;
        const int BatchOffset=BatchVolume*b;
    for (int i = 0; i < N1; i++)
    {
        for (int j = 0; j < N2; j++)
        {
            GRID_AXIS_LOOP(xIdx, xThreads, x)
            {
                GRID_AXIS_LOOP(yIdx, yThreads, y)
                {
                    GRID_AXIS_LOOP(zIdx, zThreads, z)
                    {

                        int oh = (hstride * i) + xIdx;
                        int ow = (wstride * j) + yIdx;

                        if (S2B)
                        {
                            if (oh < hSize && ow < wSize)
                            {
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] =
                                    shape[ShapeOffset + (oh * hSize * zThreads) + (ow * zThreads) + zIdx];
                            }
                            else
                            {
                                if (h_over_scan>0 && ow<wSize){
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0;
                                }
                                if (w_over_scan>0 && oh<hSize){
                                    batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0;
                                }
                            }
                        }
                        else
                        {
                            shape[ShapeOffset + (oh * hSize * zThreads) + (ow * zThreads) + zIdx] +=
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx];
                        }
                    }
                }
            }
        }
    }
}
}



//ShapetoBatch4DNCHW Does a stride shape to batch. Make sure values on receiving end are set to zero when s2b is 0


extern "C" __global__ void ShapetoBatch4DNCHW(
    const int xThreads,
    const int yThreads,
    const int zThreads,
    const int hSize,
    const int wSize,
    const int num_original_batches,
    const int BatchVolume,
    const int OriginalVol,
    const int N1,
    const int N2,
    const int hstride,
    const int wstride,
    float *shape,
    float *batch,
    const int h_over_scan,
    const int w_over_scan,
    const bool S2B)
{
    int batch0 = N2 * xThreads * yThreads * zThreads;
    int batch1 = xThreads * yThreads * zThreads;
    int batch2 = xThreads * yThreads;
    int batch3 = yThreads;
    for (int b = 0;b<num_original_batches;b++)
    {
        const int ShapeOffset = OriginalVol*b;
        const int BatchOffset=BatchVolume*b;
    for (int i = 0; i < N1; i++)
    {
        for (int j = 0; j < N2; j++)
        {
            GRID_AXIS_LOOP(xIdx, xThreads, x)
            {
                GRID_AXIS_LOOP(yIdx, yThreads, y)
                {
                    GRID_AXIS_LOOP(zIdx, zThreads, z)
                    {

                        int oh = (hstride * i) + yIdx;
                        int ow = (wstride * j) + zIdx;

                        if (S2B )
                        {
                            if (oh < hSize && ow < wSize)
                            {
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] =
                                    shape[ShapeOffset + (xIdx * wSize * hSize) + (oh * wSize) + ow];
                            }
                            else
                            {
                                if (h_over_scan>0 && ow<wSize){
                                    batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0;
                                }
                                if (w_over_scan>0 && oh<hSize){
                                    batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0; 
                                }
                               
                            }
                        }
                        else
                        {
                            shape[ShapeOffset + (xIdx * wSize * hSize) + (oh * wSize) + ow] +=
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx];
                        }
                    }
                }
            }
        }
    }
}
}


extern "C" __global__ void NearestNeighborNHWC(
    const int aligncorners,
    const int threads,
    const float *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    float *dest)
{
    GRID_LOOP_X(i, threads)
    {
        int n = i;
        int c = n % channels;
        n /= channels;
        int dest_x = n % dest_width;
        n /= dest_width;
        int dest_y = n % dest_height;
        n /= dest_height;
        const float *src_data_n = &src[n * channels * src_height * src_width];
        const int src_y = fminf((aligncorners) ? (roundf(dest_y * height_scale))
                                               : (floorf(dest_y * height_scale)),
                                src_height - 1);

        const int src_x = fminf((aligncorners) ? (roundf(dest_x * width_scale))
                                               : (floorf(dest_x * width_scale)),
                                src_width - 1);
        const int idx = (src_y * src_width + src_x) * channels + c;
        dest[i] = src_data_n[idx];
    }
}
extern "C" __global__ void NearestNeighborNCHW(
    const int aligncorners,
    const int threads,
    const float *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    float *dest)
{
    GRID_LOOP_X(i, threads)
    {
        int n = i;
        int dest_x = n % dest_width;
        n /= dest_width;
        int dest_y = n % dest_height;
        n /= dest_height;
        int c = n % channels;
        n /= channels;
        const float *src_data_n = &src[n * channels * src_height * src_width];
        const int src_y = fminf((aligncorners) ? (roundf(dest_y * height_scale))
                                               : (floorf(dest_y * height_scale)),
                                src_height - 1);

        const int src_x = fminf((aligncorners) ? (roundf(dest_x * width_scale))
                                               : (floorf(dest_x * width_scale)),
                                src_width - 1);
        const int idx = (c * src_height * src_width) + (src_y * src_width) + src_x;
        dest[i] = src_data_n[idx];
    }
}
extern "C" __global__ void NearestNeighborNCHWBack(
    const int aligncorners,
    const int threads,
    float *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    float *dest)
{
    GRID_LOOP_X(i, threads)
    {
        int n = i;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        int c = n % channels;
        n /= channels;
        float *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (c * dest_width * dest_height) + (dest_y * dest_width) + dest_x;
        atomicAdd(&src_data_n[idx], dest[i]);
    }
}
extern "C" __global__ void NearestNeighborNHWCBack(
    const int aligncorners,
    const int threads,
    float *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    float *dest)
{
    GRID_LOOP_X(i, threads)
    {
        int n = i;
        int c = n % channels;
        n /= channels;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        float *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (dest_y * dest_width + dest_x) * channels + c;
        atomicAdd(&src_data_n[idx], dest[i]);
    }
}
extern "C" __global__ void AdaGrad(const int length,
                                        float *weights,   //weights input and output
                                        float *dw,        //input and will have to set to zero
                                        float *gsum,      //storage
                                        const float rate, //input
                                        const float eps,
                                        const float dwalpha)
{ //input
    GRID_LOOP_X(cell, length)
    {
        gsum[cell] =  gsum[cell] + (dw[cell] * dw[cell]);
        weights[cell] += -(rate * dw[cell]) / (sqrtf(gsum[cell]) + eps);
        dw[cell] = dw[cell]*dwalpha; //smoothing factor.
    }
}


extern "C" __global__ void Adam(const int n,
                                     float *w,
                                     float *gsum,
                                     float *xsum,
                                     float *dw,
                                     const float rate,
                                     const float beta1,
                                     const float beta2,
                                     const float eps,
                                     const float denombeta1,
                                     const float denombeta2,
                                     const float dwalpha)
{

    GRID_LOOP_X(i, n)
    {
      
        gsum[i] = (beta1 * gsum[i]) + ((1.0 - beta1) * dw[i]);
        float gsumt = gsum[i] /denombeta1;
        xsum[i] = (beta2 * xsum[i]) + ((1.0 - beta2) * (dw[i] * dw[i]));
        float xsumt = xsum[i] / denombeta2;
        w[i] += -(rate * gsumt) / (sqrtf(xsumt) + eps);
        dw[i]=  dwalpha*dw[i]; //smoothing factor
    }
  
}

extern "C" __global__ void AdaDelta(const int length,
                                         float *weights,   //weights input and output
                                         float *gsum,      //storage
                                         float *xsum,      //storage
                                         float *dw,        //input and will have to set to zero
                                         const float rate, //input
                                         const float eps,
                                         const float ro,
                                         const float dwalpha)
{

    GRID_LOOP_X(i, length)
    {

        gsum[i] = (ro * gsum[i]) + ((1.0-ro)*dw[i] * dw[i]);
        const float dx = sqrtf((xsum[i]+eps)/(gsum[i]+eps))*dw[i];
        xsum[i]=(ro*xsum[i])+((1-ro)*dx*dx);
        weights[i] -= dx;
        dw[i] = dw[i]*dwalpha;
    }
}
/*
//This is paired with the host
extern "C" __global__ void Segment1stDim(const int start_index, const float *src, float *dst, const int size)
{
    int i = (blockIdx.y * gridDim.x * blockDim.x) + (blockIdx.x * blockDim.x) + threadIdx.x;
    int start_location = start_index * size;
    if (i < size)
    {
        dst[i] = src[start_location + i];
    }
}
//This is paired with the host
extern "C" __global__ void Segment1stDimhalf(const int start_index, const __half *src, __half *dst, const int size)
{
    int i = (blockIdx.y * gridDim.x * blockDim.x) + (blockIdx.x * blockDim.x) + threadIdx.x;
    int start_location = start_index * size;
    if (i < size)
    {
        dst[i] = src[start_location + i];
    }
}
*/
extern "C" __global__ void L1L2(
    const int length,
    float *dw,          //input and output
    const float *w,     //input needs to ba an array
    float *l1,          //output set to zero
    float *l2,          //output set to zero
    const float batch,  // should be an int but just send it as a float
    const float decay1, //input
    const float decay2)
{ //input

    GRID_LOOP_X(i, length)
    {

        atomicAdd(l1, abs(w[i]) * decay1);
        atomicAdd(l2, (w[i] * w[i] * decay2) / 2.0);
        const float gradl1 = decay1 * (w[i] > 0 ? 1 : -1);
        const float gradl2 = w[i] * decay2;
        dw[i] = (dw[i] + gradl2 + gradl1) / batch;
    }
}
//ThreshForward is kind of memory expensive, mostly because it is experimental.
//To test start the positive at random uniform numbers between .9 and 1.1
//and do the negcoefs between .01 and .2 or something along those lines.
//maybe the threshold should be between -.3 and .3 uniform number
extern "C" __global__ void ThreshForward(const int XThreads,
                                         const int batchsize,
                                         const float *x,
                                         float *y,
                                         const float *negcoefs,
                                         const float *threshhold,
                                         const float *poscoefs)
{
    for (int i=0;i<batchsize;i++)
    {
        int stride=XThreads*i;
            GRID_LOOP_X(xIdx,XThreads)
            {
                if (x[stride+xIdx]>threshhold[xIdx])
                {
                    y[stride+xIdx]=  x[stride+xIdx]*poscoefs[xIdx];
                }
                else
                {
                    y[stride+xIdx]=  negcoefs[xIdx]*x[stride+xIdx];
                }
            }
    }
}

//Backward 
// Max(x,thresh)
extern "C" __global__ void ThreshBackward(const int XThreads,
                                          const int batchsize,
                                          const float *x,
                                          float *dx,
                                          const float *dy,
                                          const float *negcoefs,
                                          float *dnegcoefs,
                                          const float *threshhold,
                                          float *dthreshhold,
                                          const float *poscoefs,
                                          float *dposcoefs)
{

    for (int i=0;i<batchsize;i++)
    {
        int stride=XThreads*i;
            GRID_LOOP_X(xIdx,XThreads)
            {
                if (x[stride+xIdx]>threshhold[xIdx])  
                {
                    dx[stride+xIdx]=  poscoefs[xIdx]*dy[stride+xIdx];
                    dposcoefs[xIdx]+=dy[xIdx]*x[stride+xIdx];

                }
                else
                {
                    dx[stride+xIdx]=  negcoefs[xIdx]*dy[stride+xIdx];
                    dnegcoefs[xIdx]+=dy[xIdx]*x[stride+xIdx];
                }
                dthreshhold[xIdx]+=dy[xIdx];
            }
    }
}

//forwardPrelu does the forward Prelu
extern "C" __global__ void PreluForward(const int XThreads,
                                        const int batchsize,
                                        const float *x,
                                        float *y,
                                        const float *coefs)
{
  
    for (int i=0;i<batchsize;i++)
    {
        int stride=XThreads*i;
            GRID_LOOP_X(xIdx,XThreads)
            {
                if (x[stride+xIdx]>0)
                {
                    y[stride+xIdx]=  x[stride+xIdx];
                }
                else
                {
                    y[stride+xIdx]=  coefs[xIdx]*x[stride+xIdx];
                }
            }
    }
   
}
//backwardPrelu does the backprop of the parametric float

extern "C" __global__ void PreluBackward(const int XThreads,
                                                          const int batchsize,
                                                          float *dx,
                                                          const float *x,
                                                          const float *dy,
                                                          const float *coefs,
                                                          float *dcoefs)
{
    for (int i=0;i<batchsize;i++)
    {
        int stride=XThreads*i;
            GRID_LOOP_X(xIdx,XThreads)
            {
                if (x[stride+xIdx]>0)
                {
                    dx[stride+xIdx]=  dy[stride+xIdx];
                }
                else
                {
                    dx[stride+xIdx]=  coefs[xIdx]*dy[stride+xIdx];
                    dcoefs[xIdx]+=dy[xIdx]*x[stride+xIdx];
                }
            }
    }
}


/*
Leaky functions
*/

extern "C" __global__ void LeakyForwardAlphaBeta(const int length,
                                             const float *x,
                                             float *y,
                                             const float coef,
                                             const float alpha,
                                              const float beta)
{

    GRID_LOOP_X(i, length)
    {
        const float previous = y[i];
        if (x[i] > 0.0)
        {
            const float current = x[i];
            y[i] = (beta*previous) + (alpha *current) ;
        }
        else
        {
              const float current = x[i]*coef;
           y[i] = (beta*previous) + (alpha *current) ;
        }
          __syncthreads();
    }
}



extern "C" __global__ void LeakyBackwardAlphaBeta(const int length,
                                              const float *x,
                                              float *dx,
                                              const float *dy,
                                              const float coef,
                                              const float alpha,
                                              const float beta)
{

    GRID_LOOP_X(i, length)
    {
        const float previous = dx[i];
        if (x[i] > 0.0)
        {
            const float current= dy[i];
            dx[i] =(beta *previous) + (current * alpha);
        }
        else
        {
            const float current= dy[i]*coef;
            dx[i] = (beta *previous) + (current * alpha);
        }
        __syncthreads();
    }
}
extern "C" __global__ void LeakyForwardAlpha(const int length,
                                             const float *x,
                                             float *y,
                                             const float coef,
                                             const float alpha)
{

    GRID_LOOP_X(i, length)
    {
        
        if (x[i] > 0.0)
        {
            y[i] = alpha *x[i];
        }
        else
        {
            const float current=x[i]*coef;
            y[i] =current * alpha;
        }
         __syncthreads();
    }
}

extern "C" __global__ void LeakyBackwardAlpha(const int length,
                                              const float *x,
                                              float *dx,
                                              const float *dy,
                                              const float coef,
                                              const float alpha)
{
 
    GRID_LOOP_X(i, length)
    {

        if (x[i] > 0.0)
        {
            dx[i] = dy[i]*alpha;
        }
        else   
        {
            const float current=dy[i]*coef;
            dx[i] = current *alpha;
        }
         __syncthreads();
    }
}


extern "C" __global__ void LeakyForward(const int length,
                                             const float *x,
                                             float *y,
                                             const float coef)
{
    GRID_LOOP_X(i, length)
    {
        if (x[i] > 0.0)
        {
            y[i] = x[i];
        }
        else
        {
            y[i] = x[i] * coef;
        }
    }
}

extern "C" __global__ void LeakyBackward(const int length,
                                              const float *x,
                                              float *dx,
                                              const float *dy,
                                              const float coef)
{

    GRID_LOOP_X(i, length)
    {

        if (x[i] > 0.0)
        {

            dx[i] = dy[i];
        }
        else
        {

            dx[i] = dy[i] * coef;
        }
    }
}

extern "C" __global__ void MSELoss(const int length, 
                            float *errors, 
                            const float *target,
                            const float *networkout, 
                            float *loss,
                            const float alpha,
                            const float beta)
{
    
    loss[0]=0;
    GRID_LOOP_X(i, length)
    {
        const float y = networkout[i] - target[i];
        errors[i] = y;
        atomicAdd(loss, (y * y) / 2);
    }

   
}

extern "C" __global__ void MSELossbyBatches(const int xthreads,const int ythreads, float *errors, const float *target, const float *networkout, float *loss)
{

    GRID_AXIS_LOOP(xIdx,xthreads,x)
    {
        const int offset=ythreads*xIdx;
            GRID_AXIS_LOOP(yIdx, ythreads,y)
            {  
             const float y = networkout[offset+yIdx] - target[offset+yIdx];
             errors[offset+yIdx] = y;
             atomicAdd(&loss[xIdx], (y * y) / 2);
            }
    }
}




extern "C" __global__ void ConcatNHWCEX(const int XThreads,
                                        const int YThreads,
                                        const int ZThreads,
                                        const int Batches,
                                        const int DestBatchVol,
                                        const int TotalDestChannels,
                                        const int DestChannelOffset,
                                        float *src,
                                        const float alpha,
                                        const int SrcBatchVol, 
                                        float *dest,
                                        const float beta,
                                        bool forward)
{
for (int i=0;i<Batches;i++){

GRID_AXIS_LOOP(idX,XThreads,x)
{
    GRID_AXIS_LOOP(idY,YThreads,y)
    {
        GRID_AXIS_LOOP(idZ,ZThreads,z)
        {
        int deststride = (i*DestBatchVol)+(idX*YThreads*TotalDestChannels)+(idY*TotalDestChannels)+DestChannelOffset+idZ;
        int srcstride = (i*SrcBatchVol)+(idX*YThreads*ZThreads)+(idY*ZThreads)+idZ;
        if (forward){
            dest[deststride]=src[srcstride]*alpha + dest[deststride]*beta;  
        }else{
            src[srcstride]=dest[deststride]*alpha + src[srcstride]*beta;
        }
         
        }
    }
}

}

}

extern "C" __global__ void ConcatNCHWEX(const int XThreads,
                                        const int Batches,
                                        const int DestBatchVol,
                                        const int DestChannelOffset,
                                        float *src,
                                        const float alpha,
                                        const int SrcBatchVol, 
                                        float *dest,
                                        const float beta,
                                        bool forward)
{
for (int i=0;i<Batches;i++){

GRID_AXIS_LOOP(idX,XThreads,x)
{

        int deststride = (i*DestBatchVol)+(DestChannelOffset+idX);
        int srcstride = (i*SrcBatchVol)+(idX);
         if (forward){
            dest[deststride]=src[srcstride]*alpha + dest[deststride]*beta;  
        }else{
            src[srcstride]=dest[deststride]*alpha + src[srcstride]*beta;
        }
         
        }
    }
}

extern "C" __global__ void ConcatNHWCEXHalf(const int XThreads,
                                        const int YThreads,
                                        const int ZThreads,
                                        const int Batches,
                                        const int DestBatchVol,
                                        const int TotalDestChannels,
                                        const int DestChannelOffset,
                                        __half *src, 
                                        const __half alpha,
                                        const int SrcBatchVol, 
                                        __half *dest,
                                        const __half beta,
                                        bool forward)
{
for (int i=0;i<Batches;i++){

GRID_AXIS_LOOP(idX,XThreads,x)
{
    GRID_AXIS_LOOP(idY,YThreads,y)
    {
        GRID_AXIS_LOOP(idZ,ZThreads,z)
        {
        int deststride = (i*DestBatchVol)+(idX*YThreads*TotalDestChannels)+(idY*TotalDestChannels)+DestChannelOffset+idZ;
        int srcstride = (i*SrcBatchVol)+(idX*YThreads*ZThreads)+(idY*ZThreads)+idZ;
         if (forward){
            dest[deststride]=__hadd(__hmul(src[srcstride],alpha), __hmul(dest[deststride],beta));  
        }else{
            src[srcstride]=__hadd(__hmul(dest[deststride],alpha), __hmul( src[srcstride],beta));
        }
         
        }
    }
}

}

}
extern "C" __global__ void ConcatNCHWEXHalf(const int XThreads,
                                            const int Batches,
                                            const int DestBatchVol,
                                            const int DestChannelOffset,
                                            __half *src, 
                                            const __half alpha,
                                            const int SrcBatchVol, 
                                            __half *dest,
                                            __const __half beta,
                                            bool forward)
{
for (int i=0;i<Batches;i++){

GRID_AXIS_LOOP(idX,XThreads,x)
{

        int deststride = (i*DestBatchVol)+(DestChannelOffset+idX);
        int srcstride = (i*SrcBatchVol)+(idX);
        if (forward){
            dest[deststride]=__hadd(__hmul(src[srcstride],alpha), __hmul(dest[deststride],beta));  
        }else{
            src[srcstride]=__hadd(__hmul(dest[deststride],alpha), __hmul( src[srcstride],beta));
        }
         
        }
    }
}

extern "C" __global__ void ConcatForwardNCHW( const int XThreads,
                                              const int Batches,
                                              const int Channels1,
                                              const int src1vol,
                                              const float *Src1,
                                              const int Channels2,
                                              const int src2vol,
                                              const float *Src2,
                                              float *dest)
{
    for (int i = 0;i<Batches;i++)
    {
        const int Stride= Batches*(src1vol+src2vol);
        const int src1batchstride=src1vol*i;
        const int src2batchstride=src2vol*i;
        for (int j=0;j<Channels1;j++)
        {
            GRID_LOOP_X(xIdx, XThreads)
            {
           dest[Stride+(j*XThreads)+xIdx]  = Src1[src1batchstride+(j*XThreads)+xIdx];
            }
        }
        for (int j=0;j<Channels2;j++){
            GRID_LOOP_X(xIdx, XThreads)
            {
           dest[Stride+(j*XThreads)+src1vol+xIdx]  = Src2[src2batchstride+(j*XThreads)+xIdx];
            }
        }
    }
}
extern "C" __global__ void ConcatBackwardNCHW( const int XThreads,
                                              const int Batches,
                                              const int Channels1,
                                              const int src1vol,
                                               float *Src1,
                                              const int Channels2,
                                              const int src2vol,
                                               float *Src2,
                                              const float *dest)
{
    for (int i = 0;i<Batches;i++)
    {
        const int Stride= Batches*(src1vol+src2vol);
        const int src1batchstride=src1vol*i;
        const int src2batchstride=src2vol*i;
        for (int j=0;j<Channels1;j++)
        {
            GRID_LOOP_X(xIdx, XThreads)
            {
                 Src1[src1batchstride+(j*XThreads)+xIdx]=  dest[Stride+(j*XThreads)+xIdx];  
            }
        }
        for (int j=0;j<Channels2;j++){
            GRID_LOOP_X(xIdx, XThreads)
            {
                Src2[src2batchstride+(j*XThreads)+xIdx]  = dest[Stride+(j*XThreads)+src1vol+xIdx];  
            }
        }
    }
}


extern "C" __global__ void ConcatForwardNCHWhalf( const int XThreads,
                                              const int Batches,
                                              const int Channels1,
                                              const int src1vol,
                                              const __half *Src1,
                                              const int Channels2,
                                              const int src2vol,
                                              const __half *Src2,
                                              __half *dest)
{
    for (int i = 0;i<Batches;i++)
    {
        const int Stride= Batches*(src1vol+src2vol);
        const int src1batchstride=src1vol*i;
        const int src2batchstride=src2vol*i;
        for (int j=0;j<Channels1;j++)
        {
            GRID_LOOP_X(xIdx, XThreads)
            {
           dest[Stride+(j*XThreads)+xIdx]  = Src1[src1batchstride+(j*XThreads)+xIdx];
            }
        }
        for (int j=0;j<Channels2;j++){
            GRID_LOOP_X(xIdx, XThreads)
            {
           dest[Stride+(j*XThreads)+src1vol+xIdx]  = Src2[src2batchstride+(j*XThreads)+xIdx];
            }
        }
    }
}
extern "C" __global__ void ConcatBackwardNCHWhalf( const int XThreads,
                                                   const int Batches,
                                                   const int Channels1,
                                                   const int src1vol,
                                               __half *Src1,
                                              const int Channels2,
                                              const int src2vol,
                                               __half *Src2,
                                              const __half *dest)
{
    for (int i = 0;i<Batches;i++)
    {
        const int Stride= Batches*(src1vol+src2vol);
        const int src1batchstride=src1vol*i;
        const int src2batchstride=src2vol*i;
        for (int j=0;j<Channels1;j++)
        {
            GRID_LOOP_X(xIdx, XThreads)
            {
                 Src1[src1batchstride+(j*XThreads)+xIdx]=  dest[Stride+(j*XThreads)+xIdx];  
            }
        }
        for (int j=0;j<Channels2;j++){
            GRID_LOOP_X(xIdx, XThreads)
            {
                Src2[src2batchstride+(j*XThreads)+xIdx]  = dest[Stride+(j*XThreads)+src1vol+xIdx];  
            }
        }
    }
}
//MakePlanarImageBatchesUint8 - for this to work all the each batch should have the same amount of channels and all the channels
//need to be the same size 
extern "C" __global__ void MakePlanarImageBatchesUint8(const int XThreads, //Should be channel size
                                                 const int Batches,
                                                 const int channelsperbatch,
                                                 const float *Srcs, //all the channels for everything.
                                                 float *dest)
{
    const int batchsize = XThreads*channelsperbatch;
    for (int i = 0;i<Batches;i++)
    {
        for (int j = 0;j<channelsperbatch;j++)
        {
            GRID_LOOP_X(xIdx, XThreads)
            {
               dest[(i*batchsize)+(j*XThreads)+xIdx]=Srcs[(j*XThreads)+xIdx];
            }
        }
    
    }
}

extern "C" __global__ void TransposeFP16(int numthreads,
               const __half *src,
               const int *buf,
               const int ndims,
               __half *dest)
{
    const int *src_strides = buf; 
    const int *dest_strides = &buf[ndims];
    const int *perm = &buf[ndims * 2];

    GRID_LOOP_X(destIdx, numthreads)
    {
        int srcIdx = 0;
        int t = destIdx;
        for (int i = 0; i < ndims; ++i)
        {
            const int ratio = t / dest_strides[i];
            t -= ratio * dest_strides[i];
            srcIdx += (ratio * src_strides[perm[i]]);
        }
        dest[destIdx] = src[srcIdx];
    }  
}




extern "C" __global__ void SwapEveryOtherFP16(
    const int n, //total batches
    const int totalbatches,
    __half *t1,
    __half *t2,
   const int start,
const int stride)
{
StartAxis(stx,x)
const int BVol = n/2;
__half2 *t1h=(half2 *)t1;
__half2 *t2h=(half2 *)t2;

            for (int i =start;i<totalbatches;i+=stride)
        {
     
            
                GRID_LOOP_X(xIdx, BVol)
                { 
                    const __half2 swapper =  t1h[(i*BVol)+(xIdx)];
                    t1h[(i*BVol) +xIdx]=t2h[(i*BVol)+xIdx];
                    t2h[(i*BVol)+xIdx]=swapper;
                }
                if (stx==0 && (n%2)){
                    const int xIdx=n-1;
                    const __half swapper =  t1[(i*n)+(xIdx)];
                    t1[(i*n) +(xIdx)]=t1[(i*n)+(xIdx)];
                    t2[(i*n)+(xIdx)]=swapper;
                }

            __syncthreads();
        }      
}
extern "C" __global__ void SwapUpperLowerFP16(
    const int xThreads, //batchsize
    const int yThreads, //batchvol
    __half *t1,
    __half *t2,
    const int t1upper,
    const int t2upper,
    const int inverse)
{
const int BVol = yThreads;
    if (t1upper>0)
    {
        GRID_AXIS_LOOP(xIdx,xThreads/2,x)
        { 
            int t2Idx;
            if (t2upper>0){
                t2Idx=xIdx;
            }else{
                t2Idx=xThreads/2 +xIdx;
            }
           
            if (xIdx < xThreads && t2Idx<xThreads)
            {
                GRID_AXIS_LOOP(yIdx, BVol,y)
                {
                    
                    const __half swapper =  t1[(xIdx*BVol)+(yIdx)];
                    t1[(xIdx*BVol) +yIdx]=t2[(t2Idx*BVol)+yIdx];
                    t2[(xIdx*BVol)+yIdx]=swapper;
                } 
            }
        }
       
    }
    else  
    {
        GRID_AXIS_LOOP(xIdx, xThreads/2,x)
        {
            const int halfIdx=(xThreads/2)+xIdx;
            int t2Idx;
            if (t2upper>0){
                t2Idx=xIdx;
            }else{
                t2Idx=halfIdx;
            }
         
            if (halfIdx < xThreads)
            {
                GRID_AXIS_LOOP(yIdx, yThreads,y)
                {
                    const __half swapper =  t1[(halfIdx*BVol)+(yIdx)];
                    t1[(halfIdx*BVol) +yIdx]=t2[(t2Idx*BVol)+yIdx];
                    t2[(halfIdx*BVol)+yIdx]=swapper;
                }
            }
        }   
    }
}


//ShapetoBatch4DNHWC Does a stride shape to batch. Make sure values on receiving end are set to zero when s2b is 0
extern "C" __global__ void ShapetoBatch4DNHWCFP16(
    const int xThreads,
    const int yThreads,
    const int zThreads,
    const int hSize,
    const int wSize,
    const int num_original_batches,
    const int BatchVolume,
    const int OriginalVol,
    const int N1,
    const int N2,
    const int hstride,
    const int wstride,
    __half *shape,
    __half *batch,
    const int h_over_scan,
    const int w_over_scan,
    const bool S2B)
{
    int batch0 = N2 * xThreads * yThreads * zThreads;
    int batch1 = xThreads * yThreads * zThreads;
    int batch2 = yThreads * zThreads;
    int batch3 = zThreads;
    for (int b = 0;b<num_original_batches;b++)
    {
        const int ShapeOffset = OriginalVol*b;
        const int BatchOffset=BatchVolume*b;
    for (int i = 0; i < N1; i++)
    {
        for (int j = 0; j < N2; j++)
        {
            GRID_AXIS_LOOP(xIdx, xThreads, x)
            {
                GRID_AXIS_LOOP(yIdx, yThreads, y)
                {
                    GRID_AXIS_LOOP(zIdx, zThreads, z)
                    {

                        int oh = (hstride * i) + xIdx;
                        int ow = (wstride * j) + yIdx;

                        if (S2B)
                        {
                            if (oh < hSize && ow < wSize)
                            {
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] =
                                    shape[ShapeOffset + (oh * hSize * zThreads) + (ow * zThreads) + zIdx];
                            }
                            else
                            {
                                if (h_over_scan>0 && ow<wSize){
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0;
                                }
                                if (w_over_scan>0 && oh<hSize){
                                    batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0;
                                }
                            }
                        }
                        else
                        {
                            shape[ShapeOffset + (oh * hSize * zThreads) + (ow * zThreads) + zIdx] +=
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx];
                        }
                    }
                }
            }
        }
    }
}
}



extern "C" __global__ void ShapetoBatch4DNCHWFP16(
    const int xThreads,
    const int yThreads,
    const int zThreads,
    const int hSize,
    const int wSize,
    const int num_original_batches,
    const int BatchVolume,
    const int OriginalVol,
    const int N1,
    const int N2,
    const int hstride,
    const int wstride,
    __half *shape,
    __half *batch,
    const int h_over_scan,
    const int w_over_scan,
    const bool S2B)
{
    int batch0 = N2 * xThreads * yThreads * zThreads;
    int batch1 = xThreads * yThreads * zThreads;
    int batch2 = xThreads * yThreads;
    int batch3 = yThreads;
    for (int b = 0;b<num_original_batches;b++)
    {
        const int ShapeOffset = OriginalVol*b;
        const int BatchOffset=BatchVolume*b;
    for (int i = 0; i < N1; i++)
    {
        for (int j = 0; j < N2; j++)
        {
            GRID_AXIS_LOOP(xIdx, xThreads, x)
            {
                GRID_AXIS_LOOP(yIdx, yThreads, y)
                {
                    GRID_AXIS_LOOP(zIdx, zThreads, z)
                    {

                        int oh = (hstride * i) + yIdx;
                        int ow = (wstride * j) + zIdx;

                        if (S2B )
                        {
                            if (oh < hSize && ow < wSize)
                            {
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] =
                                    shape[ShapeOffset + (xIdx * wSize * hSize) + (oh * wSize) + ow];
                            }
                            else
                            {
                                if (h_over_scan>0 && ow<wSize){
                                    batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0;
                                }
                                if (w_over_scan>0 && oh<hSize){
                                    batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0; 
                                }
                               
                            }
                        }
                        else
                        {
                            shape[ShapeOffset + (xIdx * wSize * hSize) + (oh * wSize) + ow] +=
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx];
                        }
                    }
                }
            }
        }
    }
}
}

extern "C" __global__ void NearestNeighborNCHWFP16(
    const int aligncorners,
    const int threads,
    const __half *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    __half *dest)
{
    GRID_LOOP_X(i, threads)
    {
        int n = i;
        int dest_x = n % dest_width;
        n /= dest_width;
        int dest_y = n % dest_height;
        n /= dest_height;
        int c = n % channels;
        n /= channels;
        const __half *src_data_n = &src[n * channels * src_height * src_width];
        const int src_y = fminf((aligncorners) ? (roundf(dest_y * height_scale))
                                               : (floorf(dest_y * height_scale)),
                                src_height - 1);

        const int src_x = fminf((aligncorners) ? (roundf(dest_x * width_scale))
                                               : (floorf(dest_x * width_scale)),
                                src_width - 1);
        const int idx = (c * src_height * src_width) + (src_y * src_width) + src_x;
        dest[i] = src_data_n[idx];
    }
}

#if __CUDA_ARCH__ >= 750 //might not work on other architectures. will probably work best with even tensors.
extern "C" __global__ void NearestNeighborNHWCBackFP16(
    const int aligncorners,
    const int threads,
    __half *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    __half *dest)
{
    GRID_LOOP_X(i, threads)
    {
        int n = i;
        int c = n % channels;
        n /= channels;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        __half *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (dest_y * dest_width + dest_x) * channels + c;

        atomicAdd(&src_data_n[idx], dest[i]);
    }
}
#else
extern "C" __global__ void NearestNeighborNHWCBackFP16(
    const int aligncorners,
    const int threads,
    __half *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    __half *dest)
{
    
      const __half zer0= __float2half(0.0);
    GRID_LOOP_X(i, threads-1) //minus one because I do a conversion to half2 wich is 32bit to do the atomic add and don't want to run into space outside of array 
      
    {
        int n = i;
        int c = n % channels;
        n /= channels;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        __half *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (dest_y * dest_width + dest_x) * channels + c;
        const __half2 dsth2 = __halves2half2(dest[i],zer0);   // This should give us the value half2[dest,0]
        void *vdptr=(void*)(&src_data_n[idx]);  //I don't know if I need to do this, but I work with go a lot and wanted to make sure it was going to step correctly
        __half2 *srch2hack = (__half2*)(vdptr); //Here say the void pointer address into srch2hack
        atomicAdd(srch2hack,dsth2); // this should be (src_data_n[idx]+dest[i], src_data_n[idx+1]+0)  //had to do threads -1 so in the last part we don't overstep the bounds
    }
    //This last part is to do the last value in dest.
     int n = threads-1;
       int c = n % channels;
        n /= channels;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        __half *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (dest_y * dest_width + dest_x) * channels + c;
         src_data_n[idx] = __hadd(src_data_n[idx], dest[threads-1]);
}

#endif

extern "C" __global__ void NearestNeighborNHWCFP16(
    const int aligncorners,
    const int threads,
    const __half *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    __half *dest)
{
    
    GRID_LOOP_X(i, threads)
    {
        int n = i;
        int c = n % channels;
        n /= channels;
        int dest_x = n % dest_width;
        n /= dest_width;
        int dest_y = n % dest_height;
        n /= dest_height;
        const __half *src_data_n = &src[n * channels * src_height * src_width];
        const int src_y = fminf((aligncorners) ? (roundf(dest_y * height_scale))
                                               : (floorf(dest_y * height_scale)),
                                src_height - 1);

        const int src_x = fminf((aligncorners) ? (roundf(dest_x * width_scale))
                                               : (floorf(dest_x * width_scale)),
                                src_width - 1);                 
        const int idx = (src_y * src_width + src_x) * channels + c;
        dest[i] = src_data_n[idx];
    }
}


#if __CUDA_ARCH__ >= 750 //might not work on other architectures. will probably work best with even tensors.
extern "C" __global__ void NearestNeighborNCHWBackFP16(
    const int aligncorners,
    const int threads,
    __half *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    __half *dest)
{
    GRID_LOOP_X(i, threads)
    {
        int n = i;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        int c = n % channels;
        n /= channels;
        __half *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (c * dest_width * dest_height) + (dest_y * dest_width) + dest_x;
        atomicAdd(&src_data_n[idx], dest[i]);
    }
}
#else

//Might not work with archs that are not 7.5.. but might work best with even tensors.
extern "C" __global__ void NearestNeighborNCHWBackFP16(
    const int aligncorners,
    const int threads,
    __half *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    __half *dest)
{
    
      const __half zer0= __float2half(0.0);
    GRID_LOOP_X(i, threads-1) //minus one because I do a conversion to half2 wich is 32bit to do the atomic add and don't want to run into space outside of array 
      
    {

        int n = i;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        int c = n % channels;
        n /= channels;
        __half *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (c * dest_width * dest_height) + (dest_y * dest_width) + dest_x;
        const __half2 dsth2 = __halves2half2(dest[i],zer0);   // This should give us the value half2[dest,0]
        void *vdptr=(void*)(&src_data_n[idx]);  //I don't know if I need to do this, but I work with go a lot and wanted to make sure it was going to step correctly
        __half2 *srch2hack = (__half2*)(vdptr); //Here say the void pointer address into srch2hack
        atomicAdd(srch2hack,dsth2); // this should be (src_data_n[idx]+dest[i], src_data_n[idx+1]+0)  //had to do threads -1 so in the last part we don't overstep the bounds
    }
    //This last part is to do the last value in dest.
     int n = threads-1;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        int c = n % channels;
        n /= channels;
        __half *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (c * dest_width * dest_height) + (dest_y * dest_width) + dest_x;
      src_data_n[idx] = __hadd(src_data_n[idx], dest[threads-1]);
}
#endif
extern "C" __global__ void AdaGradFP16(const int n,
                                        __half *w,   //w input and output
                                        __half *dw,        //input and will have to set to zero
                                        __half *gsum,      //storage
                                        const __half rate, //input
                                        const __half eps,
                                        const __half dwalpha)
{ //input
    StartAxis(stx,x)
    int n2=n/2;
    __half2 *w2=(__half2*)w,*dw2=(__half2*)dw,*gsum2=(__half2*)gsum;
    
    const __half2 rate2=__halves2half2(rate,rate);
    const __half2 eps2=__halves2half2(eps,eps);
    const __half2 dwalpha2=__halves2half2(dwalpha,dwalpha);
    GRID_LOOP_X(i, n2)
    {
        __half2 holder = gsum2[i];
        gsum2[i] = __hfma2(dw2[i],dw2[i],holder);
        w2[i] = __hadd2(-__h2div((__hmul2(rate2,dw2[i])) , (__hadd2(h2sqrt(gsum2[i]), eps2))),w2[i]);
        dw2[i] =__hmul2(dw2[i],dwalpha2);
    }
    if (stx==0 && (n%2)){
        __half holder = gsum[n-1];
        gsum[n-1] = __hfma(dw[n-1],dw[n-1],holder);
        w[n-1] = -__hdiv((__hmul(rate,dw[n-1])) , (__hadd(hsqrt(gsum[n-1]), eps)));
        dw[n-1] =__hmul(dw[n-1],dwalpha);
    }
}


//Need to fix this.
extern "C" __global__ void AdamFP16(const int n,
                                     __half *w,
                                     __half *gsum,
                                     __half *xsum,
                                     __half *dw,
                                     const __half rate,
                                     const __half beta1,
                                     const __half beta2,
                                     const __half eps,
                                     const __half denombeta1,
                                     const __half denombeta2,
                                     const __half dwalpha)
{
    int n2=n/2;
    __half2 *w2=(__half2*)w,*dw2=(__half2*)dw,*gsum2=(__half2*)gsum,*xsum2=(__half2*)xsum;
    const __half2 rate2=__halves2half2(rate,rate);
    const __half2 eps2=__halves2half2(eps,eps);
    const __half2 dwalpha2=__halves2half2(dwalpha,dwalpha);
    const __half2 beta12=__halves2half2(beta1,beta1);
    const __half2 beta22=__halves2half2(beta2,beta2);
     const __half one1 = __float2half(1.0);
  const __half2 one2=__halves2half2(one1,one1);
    StartAxis(stx,x)
    GRID_LOOP_X(i, n2)
    {
      gsum2[i] =__hfma2(__hsub2(one2,beta12),dw2[i],__hmul2(beta12,gsum2[i]));
     __half2 gsumt = __h2div(gsum2[i] ,__halves2half2(denombeta1,denombeta1));
      xsum2[i] = __hfma2(beta22 , xsum2[i], __hmul2(__hsub2(one2, beta22), __hmul2(dw2[i] , dw2[i])));
     __half2 xsumt = __h2div(xsum2[i] , __halves2half2(denombeta2,denombeta2));
     w2[i]=__hsub2(w2[i],__h2div(__hmul2(rate2,gsumt),__hadd2(h2sqrt(xsumt),eps2)));
     dw2[i]=  __hmul2(dwalpha2,dw2[i]);
    }
 
        if (stx==0 && (n%2)){
            const int i = n-1;
             gsum[i] =__hfma(__hsub(one1,beta1),dw[i],__hmul(beta1,gsum[i]));
            __half gsumt = __hdiv(gsum[i] ,denombeta1);
              xsum[i] = __hfma(beta2 , xsum[i], __hmul(__hsub(one1, beta2), __hmul(dw[i] , dw[i])));
             __half xsumt = __hdiv(xsum[i] , denombeta2);
             w[i]=__hsub(w[i],__hdiv(__hmul(rate,gsumt),__hadd(hsqrt(xsumt),eps)));
            dw[i]=  __hmul(dwalpha,dw[i]);
      }
}


extern "C" __global__ void AdaDeltaFP16(const int n,
                                         __half *w,   //weights input and output
                                         __half *gsum,      //storage
                                         __half *xsum,      //storage
                                         __half *dw,        //input and will have to set to zero
                                         const __half rate, //input
                                         const __half eps,
                                          const __half ro,
                                         const __half dwalpha)
{
    StartAxis(stx,x)
    int n2=n/2;
     __half2 *w2=(__half2*)w,*dw2=(__half2*)dw,*gsum2=(__half2*)gsum,*xsum2=(__half2*)xsum;
    const __half2 rate2=__halves2half2(rate,rate);
    const __half2 eps2=__halves2half2(eps,eps);
    const __half2 ro2=__halves2half2(ro,ro);
   const __half one1 = __float2half(1.0);
  const __half2 one2=__halves2half2(one1,one1);
    const __half2 dwalpha2=__halves2half2(dwalpha,dwalpha);
    GRID_LOOP_X(i, n2)
    {
       gsum2[i]= __hfma2(__hsub2(one2,ro2),__hmul2(dw2[i],dw2[i]),__hmul2(ro2,gsum2[i]));
       const __half2 dx2= __hmul2(h2sqrt(__h2div(__hadd2(xsum2[i],eps2),__hadd2(gsum2[i],eps2))),dw2[i]);
       xsum2[i]= __hfma2(__hsub2(one2,ro2),__hmul2(dx2,dx2),__hmul2(ro2,xsum2[i]));
       w2[i] =__hsub2(w2[i],dx2);
       dw2[i] =  __hmul2(dw2[i],dwalpha2);
    }
  
    if (stx ==0 &&(n%2)){
       int i = n-1;
       gsum[i]= __hfma(__hsub(one1,ro),__hmul(dw[i],dw[i]),__hmul(ro,gsum[i]));
       const __half dx= __hmul(hsqrt(__hdiv(__hadd(xsum[i],eps),__hadd(gsum[i],eps))),dw[i]);
       xsum[i]= __hfma(__hsub(one1,ro),__hmul(dx,dx),__hmul(ro,xsum[i]));
       w[i] =__hsub(w[i],dx);
       dw[i] =  __hmul(dw[i],dwalpha);
    }
}

#if __CUDA_ARCH__ >= 750
extern "C" __global__ void L1L2FP16(
    const int length,
    __half *dw,          //input and output
    const __half *w,     //input needs to ba an array
    __half *l1,          //output set to zero
    __half *l2,          //output set to zero
    const __half batch,  // should be an int but just send it as a float
    const __half decay1, //input
    const __half decay2)
{ //input
  const __half one1 = __float2half(1.0);
    const __half zero0 = __float2half(0);
    GRID_LOOP_X(i, length)
    {
        __half abs = w[i];
        if (__hlt(abs,zero0)){
            abs=-abs;
        }
        //atomicAdd(l1, abs(w[i]) * decay1);
        atomicAdd(l1,__hmul(abs,decay1));
        //atomicAdd(l2, (w[i] * w[i] * decay2) / 2.0);
        atomicAdd(l2, __hdiv(__hmul(__hmul(w[i] , w[i]) , decay2) , 2.0));
        //const float gradl1 = decay1 * (w[i] > 0 ? 1 : -1);
        const __half gradl1 = __hmul(decay1, (__hgt(w[i],zero0) ? one1 : -one1));
        //const float gradl2 = w[i] * decay2;
        const __half gradl2 = __hmul(w[i] ,decay2);
        //dw[i] = (dw[i] + gradl2 + gradl1) / batch;     
        dw[i] = __hdiv(__hadd(__hadd(dw[i], gradl2) , gradl1) , batch);
    }
}

#else
extern "C" __global__ void L1L2FP16(
    const int length,
    __half *dw,          //input and output
    const __half *w,     //input needs to ba an array
    __half *l1,          //output set to zero
    __half *l2,          //output set to zero
    const __half batch,  // should be an int but just send it as a float
    const __half decay1, //input
    const __half decay2)
{ //input
  const __half one1 = __float2half(1.0);
    const __half zero0 = __float2half(0);
    __shared__ __half2 *l1l2h2;
    __half2 *l1h2=&l1l2h2[0];
     __half2 *l2h2=&l1l2h2[1];
    GRID_LOOP_X(i, length)
    {
        __half abs = w[i];
        if (__hlt(abs,zero0)){
            abs=-abs;
        }
        //atomicAdd(l1, abs(w[i]) * decay1);
         const __half2 result= __halves2half2( __hmul(abs,decay1),zero0);
        atomicAdd(l1h2,result);
        //atomicAdd(l2, (w[i] * w[i] * decay2) / 2.0);
               const __half2 result2= __halves2half2(__hdiv(__hmul(__hmul(w[i] , w[i]) , decay2) , 2.0),zero0);
        atomicAdd(l2h2,result2 );
        //const float gradl1 = decay1 * (w[i] > 0 ? 1 : -1);
        const __half gradl1 = __hmul(decay1, (__hgt(w[i],zero0) ? one1 : -one1));
        //const float gradl2 = w[i] * decay2;
        const __half gradl2 = __hmul(w[i] ,decay2);
        //dw[i] = (dw[i] + gradl2 + gradl1) / batch;     
        dw[i] = __hdiv(__hadd(__hadd(dw[i], gradl2) , gradl1) , batch);
    }
    l1[0]=__low2half(l1h2[0]);
     l2[0]=__low2half(l2h2[0]);
}

#endif



extern "C" __global__ void ThreshForwardFP16(const int XThreads,
                                         const int batchsize,
                                         const __half *x,
                                         __half *y,
                                         const __half *negcoefs,
                                         const __half *threshhold,
                                         const __half *poscoefs)
{
    for (int i=0;i<batchsize;i++)
    {
        int stride=XThreads*i;
            GRID_LOOP_X(xIdx,XThreads)
            {
                if (__hgt(x[stride+xIdx],threshhold[xIdx]))
                {
                    y[stride+xIdx]=  __hmul(x[stride+xIdx],poscoefs[xIdx]);
                }
                else
                {
                    y[stride+xIdx]=   __hmul(negcoefs[xIdx],x[stride+xIdx]);
                }
            }
    }
}


extern "C" __global__ void ThreshBackwardFP16(const int XThreads,
                                          const int batchsize,
                                          const __half *x,
                                          __half *dx,
                                          const __half *dy,
                                          const __half *negcoefs,
                                          __half *dnegcoefs,
                                          const __half *threshhold,
                                          const __half *poscoefs,
                                          __half *dposcoefs)
{
    for (int i=0;i<batchsize;i++)
    {
        int stride=XThreads*i;
            GRID_LOOP_X(xIdx,XThreads)
            {
                if (__hgt(x[stride+xIdx],threshhold[xIdx]))
                {
                 //  dx[stride+xIdx]=  poscoefs[xIdx]*dy[stride+xIdx];
                 dx[stride+xIdx]=__hmul(dy[stride+xIdx],poscoefs[xIdx]);
                 // dposcoefs[xIdx]+=dy[xIdx]*x[stride+xIdx];
                 dposcoefs[xIdx]=__hfma(dy[xIdx],x[stride+xIdx],dposcoefs[xIdx]);
                }
                else
                {
                  // dx[stride+xIdx]=  negcoefs[xIdx]*dy[stride+xIdx];
                  dx[stride+xIdx]= __hmul(dy[stride+xIdx],negcoefs[xIdx]);
                  // dnegcoefs[xIdx]+=dy[xIdx]*x[stride+xIdx];
                  dnegcoefs[xIdx]=__hfma(dy[xIdx],x[stride+xIdx],dnegcoefs[xIdx]);
                }
            }
    }
}

extern "C" __global__ void PreluForwardFP16(const int XThreads,
                                        const int batchsize,
                                        const __half *x,
                                        __half *y,
                                        const __half *coefs)
{
  
    for (int i=0;i<batchsize;i++)
    {
        int stride=XThreads*i;
            GRID_LOOP_X(xIdx,XThreads)
            {
                if (__hgt(x[stride+xIdx],0))
                {
                    y[stride+xIdx]=  x[stride+xIdx];
                }
                else
                {
                    y[stride+xIdx]=  __hmul(coefs[xIdx],x[stride+xIdx]);
                }
            }
    }
   
}    

extern "C" __global__ void PreluBackwardFP16(const int XThreads,
                                                          const int batchsize,
                                                          __half *dx,
                                                          const __half *x,
                                                          const __half *dy,
                                                          const __half *coefs,
                                                          __half *dcoefs)
{
        const __half zero0 = __float2half(0);
    for (int i=0;i<batchsize;i++)
    {
        int stride=XThreads*i;
            GRID_LOOP_X(xIdx,XThreads)
            {
               if (__hgt(x[stride+xIdx],zero0))
                {
                    dx[stride+xIdx]=  dy[stride+xIdx];
                }
                else
                {
                 //  dx[stride+xIdx]=  coefs[xIdx]*dy[stride+xIdx];
                  dx[stride+xIdx]=  __hmul(coefs[xIdx],dy[stride+xIdx]);
                 // dcoefs[xIdx]+=dy[xIdx]*x[stride+xIdx];
                 dcoefs[xIdx]=__hfma(dy[xIdx],x[stride+xIdx],dcoefs[xIdx]);
                }
            }
    }
}
extern "C" __global__ void LeakyForwardAlphaBetaFP16(const int length,
                                             const __half *x,
                                             __half *y,
                                             const __half coef,
                                             const __half alpha,
                                              const __half beta)
{
        const __half zero0 = __float2half(0);
    GRID_LOOP_X(i, length)
    {
       
      if (__hgt(x[i],zero0))
        {
            // y[i] = (beta*y[i]) + (alpha *x[i]) ;
            y[i]=__hadd(__hmul(beta,y[i]),__hmul(alpha,x[i]));
        }
        else
        {
         //y[i] = (beta*previous) + (alpha *x[i]*coef);
         y[i]=__hadd(__hmul(beta,y[i]),__hmul(alpha,__hmul(x[i],coef)));
        }
          __syncthreads();
    }
}
extern "C" __global__ void LeakyBackwardAlphaBetaFP16(const int length,
                                              const __half *x,
                                              __half *dx,
                                              const __half *dy,
                                              const __half coef,
                                              const __half alpha,
                                              const __half beta)
{
    const __half zero0 = __float2half(0);
    GRID_LOOP_X(i, length)
    {
  
        if (__hgt(x[i],zero0))
        {
             // dx[i] =(beta *dx[i]) + (dy[i] * alpha);
              dx[i]=__hadd(__hmul(beta,dy[i]),__hmul(alpha,dx[i]));
        }
        else
        {
             // dx[i] = (beta *dx[i]) + (dy[i]*coef * alpha);
             dx[i]=__hadd(__hmul(beta,dx[i]),__hmul(alpha,__hmul(dy[i],coef)));
        }
        __syncthreads();
    }
}
extern "C" __global__ void LeakyForwardAlphaFP16(const int length,
                                             const __half *x,
                                             __half *y,
                                             const __half coef,
                                             const __half alpha)
{
    const __half zero0 = __float2half(0);
    GRID_LOOP_X(i, length)
    {
        
      if (__hgt(x[i],zero0))
        {
            y[i] = __hmul(alpha ,x[i]);
        }
        else
        {
        
            y[i] =__hmul(__hmul(x[i],coef) , alpha);
        }
         __syncthreads();
    }
}


extern "C" __global__ void LeakyBackwardAlphaFP16(const int length,
                                              const __half *x,
                                              __half *dx,
                                              const __half *dy,
                                              const __half coef,
                                              const __half alpha)
{
        const __half zero0 = __float2half(0);
 
    GRID_LOOP_X(i, length)
    {

        if  (__hgt(x[i],zero0))
        {
           // dx[i] = dy[i]*alpha;
            dx[i] = __hmul(alpha ,dy[i]);
        }
        else
        {
             // dx[i] = dy[i]*coef *alpha;
             dx[i] =__hmul(__hmul(dy[i],coef) , alpha);
        }
         __syncthreads();
    }
}

extern "C" __global__ void LeakyForwardFP16(const int length,
                                             const __half *x,
                                             __half *y,
                                             const __half coef)
{
        const __half zero0 = __float2half(0);
    GRID_LOOP_X(i, length)
    {
       if  (__hgt(x[i],zero0))
        {
            y[i] = x[i];
        }
        else
        {
         //   y[i] = x[i] * coef;
       y[i]= __hmul( x[i] , coef);
        }
    }
}

extern "C" __global__ void LeakyBackwardFP16(const int length,
                                              const __half *x,
                                              __half *dx,
                                              const __half *dy,
                                              const __half coef)
{
    const __half zero0 = __float2half(0);
    GRID_LOOP_X(i, length)
    {

         if  (__hgt(x[i],zero0))
        {
            dx[i] = dy[i];
        }
        else
        {
//       dx[i] = dy[i] * coef;
         dx[i]= __hmul( dy[i] , coef);
        }
    }
}


#if __CUDA_ARCH__ >= 750
extern "C" __global__ void MSELossbyBatchesFP16(const int xthreads,
const int ythreads,
 __half *errors, 
 const __half *target, 
 const __half *networkout, 
 __half *loss)
{
  const __half htwo= __float2half(2.0);
    GRID_AXIS_LOOP(xIdx,xthreads,x)
    {
        const int i=ythreads*xIdx;
            GRID_AXIS_LOOP(yIdx, ythreads,y)
            {  
                const __half y = __hsub(networkout[i] , target[i]);
        errors[i] = y;
             atomicAdd(&loss[xIdx], __hdiv(__hmul(y , y) , htwo));
            }
    }
}
extern "C" __global__ void MSELossFP16(const int n, 
                            __half *errors, 
                            const __half *target,
                            const __half *networkout, 
                            __half *loss,
                            const __half alpha,
                            const __half beta)
{
    StartAxis(stx,x)
    int n2=n/2;
     __half2 *errors2=(__half2*)errors, *target2=(__half2*)target, *networkout2=(__half2*)networkout, *loss2=(__half2*)loss;
 //  const __half2 alpha2=__halves2half2(alpha), beta2=__halves2half2(beta);
    const __half2 htwo2=__halves2half2(__float2half(2.0),__float2half(2.0));
     const __half htwo= __float2half(2.0);
    loss[0]=0;
    GRID_LOOP_X(i, n2)
    {
        const __half2 y = __hsub2(networkout2[i] , target2[i]);
        errors2[i] = y;
        atomicAdd(loss2, __h2div(__hmul2(y , y) ,htwo2));
    }
    if (stx==0 && (n%2)){
       const int i=n-1;
        const __half y = __hsub(networkout[i] , target[i]);
        errors[i] = y;
        atomicAdd(loss, __hdiv(__hmul(y , y) , htwo));
    }
      

   
}
#else  
extern "C" __global__ void MSELossbyBatchesFP16(
const int xthreads,
const int batches,
 __half2 *errors, 
 const __half2 *target, 
 const __half2 *networkout, 
 __half *loss)
{
  const __half htwo= __float2half(2.0);
  const __half2 htwo2 =__halves2half2(htwo,htwo);
  const int n=xthreads/2;
  __shared__ __half2 *loss2;
  for (int i=0; i<batches;i++){
      loss2[i]=__floats2half2_rn(0.0,0.0);
 GRID_AXIS_LOOP(xIdx,n,x)
    {
       const __half2 y = __hsub2(networkout[i*n+xIdx] , target[i*n+xIdx]);
       errors[i] = y;
       atomicAdd(&loss2[i], __h2div(__hmul2(y , y) , htwo2));
    }
  loss[i]=__hadd(__low2half(loss2[i]),__high2half(loss2[i]));
  }
   
}
extern "C" __global__ void MSELossFP16(const int n, 
                            __half2 *errors, 
                            const __half2 *target,
                            const __half2 *networkout, 
                            __half *loss,
                            const __half alpha,
                            const __half beta)
{
//    StartAxis(stx,x)
    int n2=n/2;
 //  const __half2 alpha2=__halves2half2(alpha), beta2=__halves2half2(beta);
    const __half2 htwo2=__halves2half2(__float2half(2.0),__float2half(2.0));
   //  const __half htwo= __float2half(2.0);
      __shared__ __half2 *loss2;
      loss2[0]= __halves2half2(__float2half(0.0),__float2half(0.0));
   
    GRID_LOOP_X(i, n2)
    {
        const __half2 y = __hsub2(networkout[i] , target[i]);
        errors[i] = y;
        atomicAdd(loss2, __h2div(__hmul2(y , y) ,htwo2));
    }
    loss[0]=__hadd(__low2half(loss2[0]),__high2half(loss2[0]));
    
}
#endif
/*
extern "C" __global__ void SoftMaxErrAndLoss(const int xthreads, const int ntargets, const float *target, const float *softmaxoutput, float *loss, float * inputerrors){
    const float fntargets=(float)(ntargets);
    GRID_LOOP_X(xIdx,xthreads){
        if (target[xIdx]>0){
            atomicAdd(&loss[xIdx],-log10(softmaxoutput[xIdx]/fntargets));
        }
    }
}
*/
extern "C" __global__ void SoftMaxAverageLoss(const int xthreads, const int ntargets, const float *target, const float *softmaxoutput, float *loss){
     const float fntargets = (float)(ntargets);
    GRID_LOOP_X(xIdx,xthreads){
        if (target[xIdx]>0){
            atomicAdd(&loss[0],-log10(softmaxoutput[xIdx])/fntargets);
        }
    }
}
/*
extern "C" __global__ void SoftMaxLossPerBatch(const int xthreads,const int ythreads, const int ntargetsperbatch,  const float *target, const float *softmaxoutput,float *loss)
{
    const float npbtargs = (float)(ntargetsperbatch);
    GRID_AXIS_LOOP(xIdx,xthreads,x)
    {       
            const int offset=ythreads*xIdx;
            GRID_AXIS_LOOP(yIdx, ythreads,y)
            {  
                if (target[offset+yIdx]>0){
                    atomicAdd(&loss[xIdx],-log10(softmaxoutput[offset+yIdx])/npbtargs);
                }
            
            }
    }
}
*/

/*
#if __CUDA_ARCH__ >= 750
extern "C" __global__ void MSELossbyBatchesFP16(const int xthreads,
const int ythreads,
 __half *errors, 
 const __half *target, 
 const __half *networkout, 
 __half *loss)
{
  const __half htwo= __float2half(2.0);
    GRID_AXIS_LOOP(xIdx,xthreads,x)
    {
        const int i=ythreads*xIdx;
            GRID_AXIS_LOOP(yIdx, ythreads,y)
            {  
                const __half y = __hsub(networkout[i] , target[i]);
        errors[i] = y;
             atomicAdd(&loss[xIdx], __hdiv(__hmul(y , y) , htwo));
            }
    }
}
extern "C" __global__ void MSELossFP16(const int n, 
                            __half *errors, 
                            const __half *target,
                            const __half *networkout, 
                            __half *loss,
                            const __half alpha,
                            const __half beta)
{
    StartAxis(stx,x)
    int n2=n/2;
     __half2 *errors2=(__half2*)errors, *target2=(__half2*)target, *networkout2=(__half2*)networkout, *loss2=(__half2*)loss;
 //  const __half2 alpha2=__halves2half2(alpha), beta2=__halves2half2(beta);
    const __half2 htwo2=__halves2half2(__float2half(2.0),__float2half(2.0));
     const __half htwo= __float2half(2.0);
    loss[0]=0;
    GRID_LOOP_X(i, n2)
    {
        const __half2 y = __hsub2(networkout2[i] , target2[i]);
        errors2[i] = y;
        atomicAdd(loss2, __h2div(__hmul2(y , y) ,htwo2));
    }
    if (stx==0 && (n%2)){
       const int i=n-1;
        const __half y = __hsub(networkout[i] , target[i]);
        errors[i] = y;
        atomicAdd(loss, __hdiv(__hmul(y , y) , htwo));
    }
      

   
}
#else  
extern "C" __global__ void MSELossbyBatchesFP16(
const int xthreads,
const int batches,
 __half2 *errors, 
 const __half2 *target, 
 const __half2 *networkout, 
 __half *loss)
{
  const __half htwo= __float2half(2.0);
  const __half2 htwo2 =__halves2half2(htwo,htwo);
  const int n=xthreads/2;
  __shared__ __half2 *loss2;
  for (int i=0; i<batches;i++){
      loss2[i]=__floats2half2_rn(0.0,0.0);
 GRID_AXIS_LOOP(xIdx,n,x)
    {
       const __half2 y = __hsub2(networkout[i*n+xIdx] , target[i*n+xIdx]);
       errors[i] = y;
       atomicAdd(&loss2[i], __h2div(__hmul2(y , y) , htwo2));
    }
  loss[i]=__hadd(__low2half(loss2[i]),__high2half(loss2[i]));
  }
   
}
extern "C" __global__ void MSELossFP16(const int n, 
                            __half2 *errors, 
                            const __half2 *target,
                            const __half2 *networkout, 
                            __half *loss,
                            const __half alpha,
                            const __half beta)
{
//    StartAxis(stx,x)
    int n2=n/2;
 //  const __half2 alpha2=__halves2half2(alpha), beta2=__halves2half2(beta);
    const __half2 htwo2=__halves2half2(__float2half(2.0),__float2half(2.0));
   //  const __half htwo= __float2half(2.0);
      __shared__ __half2 *loss2;
      loss2[0]= __halves2half2(__float2half(0.0),__float2half(0.0));
   
    GRID_LOOP_X(i, n2)
    {
        const __half2 y = __hsub2(networkout[i] , target[i]);
        errors[i] = y;
        atomicAdd(loss2, __h2div(__hmul2(y , y) ,htwo2));
    }
    loss[0]=__hadd(__low2half(loss2[0]),__high2half(loss2[0]));
    
}
#endif
*/